#include "hip/hip_runtime.h"
%%writefile testdims.cu
#include <cstdlib>
#include <stdio.h>
#include <sys/time.h>
#include <math.h>
#include "wave.h"
​
__global__
void gerarFrames(int width, int frames, unsigned char* pic){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int frame = 0; frame < frames; frame++) {
    for (int row = index; row < width; row += stride) {
      for (int col = 0; col < width; col++) {
        float fx = col - 1024/2;
        float fy = row - 1024/2;
        float d = sqrtf( fx * fx + fy * fy );
        unsigned char color = (unsigned char) (160.0f + 127.0f *
                                          cos(d/10.0f - frame/7.0f) /
                                          (d/50.0f + 1.0f));
        pic[frame * width * width + row * width + col] = (unsigned char) color;
      }
    }
  }
}
​
int main(int argc, char *argv[])
{
​
  if (argc != 3) {
    printf("ERRO: usar %s largura_frame num_frames\n", argv[0]);
    exit(-1);
  }
  int width = atoi(argv[1]);
  if (width < 100) {
    printf("ERRO: largura_frame deve ser maior igual a 100\n");
    exit(-1);
  }
  int frames = atoi(argv[2]);
  if (frames < 1) {
    printf("ERRO: num_frames deve ser pelo menos 1\n");
    exit(-1);
  }
  printf("Computando %d frames de dimensão %d por %d\n", frames, width, width);
​
  unsigned char* pic;
  hipMallocManaged(&pic, frames*width*width*sizeof(unsigned char));
    
  // inicia tempo
  timeval start, end;
  gettimeofday(&start, NULL);
  int blockSize = 512;
  int numBlocks = (width + blockSize) / blockSize;
  gerarFrames<<<numBlocks, blockSize>>>(width,frames, pic);
​
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
​
  // termina o tempo
  gettimeofday(&end, NULL);
  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  printf("compute time: %.4f s\n", runtime);
​
  hipFree(pic);
  return 0;
}